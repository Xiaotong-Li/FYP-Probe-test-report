#include "hip/hip_runtime.h"
/*! \file cuTFMv4007.cu 
 * \brief ZISC main entry point file
 * this file links cuTFM, coeffGen and FMCSim into a single executable
 */

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == ZISC cuTFM - main entry point file
// == version v4007
// == Author: Jerzy Dziewierz
// == Centre for Ultrasonic Engineering, University of Strahclyde
// == 2008-2012
// == Internal use only, do not release
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 


// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=
// == Main page for the Doxygen automatically generated documentation
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=

/*! \mainpage Welcome to the ZISK
 *
 * \section intro_sec What is this?
 * This is an automatically generated document. It contains a readable form of comments that are normally placed in the source code.
 * 
 * There won't be much basic explanations here, but rather advanced for people who are actually interested in hacking the source code. No free lunch!
 *
 * Please see the file list for basic introduction to what's in the box.
 * \section history History of the name
 * 
 * The original name was supposed to be Strais (Strathclyde Imaging System) but that sounds too much like "strata" (loss) in Polish . . . so I had to quickly come up with something else. Now "ZISK" sounds nearly like "gain" in Polish, that's a clearly a better name!
 *
 * Jerzy Dziewierz, University of Strathclyde
 * Copyright 2009-2013
 *
 */

// cuTFM v4007.cu
// entry point file for cuTFM system

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=
// == system wide libraries
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=

// Matlab libraries
#include "matrix.h"
#include "mex.h"

// C libraries
#include "omp.h"
#include "math.h"
#include "float.h"
#include "limits.h"


// CUDA libraries
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_math_constants.h"

// THRUST libraries
#include <thrust/device_ptr.h>
#include <thrust/fill.h>




// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=
// == Build related constants
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=


// debug option: force persistent_deviceCount to 1
// #define DEBUG_FORCESINGLEGPU       

// max no. of tx/rx elements, this limits coeff table size
#define max_tx 170
#define max_image_size 4*4096
#define max_devicecount 8
#define FIRSTDEVICE 0 // can be used to limit number of devices used

// there is 5 coefficients per tx-line pair, 5 or 8 is a good number
// 5 for higher performance, 7 for higher accuracy, up to 16 are supported
// note that tests show that it's the simplex descent fminsearch that is inaccurate and unstable, 
// using more than 5 coefficients will rarely result in actually better timing accuracy
// NOTE: less than 5 coeffs not supported (could be if need arises)
#define COEFF_SIZE 5

// if this is defined, linear interpolation is used for picking up values from FMC.
// otherwise the propagation time is truncated to nearest lower integer sample (that's faster)
#define USE_FMC_LINEAR_INTERPOLATION
// #define USE_FMC_NEAREST_SAMPLE

// if this is defined, an older, slower version of fminsearch is used
// if this is undefined, newer, GPU-optimised version of fminsearch is used
#define USE_CLASSIC_FMINSEARCH 

// if this is defined, CPU TimePoint generation path is used
// if this is undefined, GPU path is used
 #define USE_CPU_TimePointGenerator

// allowable fit error for the coeff generator
#define COEFFGEN_ALLOWABLE_FITERROR 1e-8

// coeffgen_nTimePointsPerLine constant that says how many time points to generate
// note that for 5 coeffs, 10+9 would be enough
// for 8 coeffs, 17+16 are required 
// this means 17 are used for coeff fit and 16 are used to verify the error bound
// Note that the polyfit17x* require 17 points and will not work with different number of points
#define COEFFGEN_nTimePointsPerToFit 17
#define COEFFGEN_nTimePointsPerToCheck 16

// note the fit and eval points are interleaved : F E F E F
// F is best to be odd because this gives better stability for polyfit* routines
// E can be F-1 and currently is, see comments near 'fitcheckZ', 'fitinput'
// Total time points per line is F+E
// will drop silent errors if this is not the case
#define COEFFGEN_nTimePointsPerLine 33

// if the RESPECT_DIRCOSLIMIT, there is additional code in the RenderTFMImage that disables accumulation of FMC sample if the cosine of the angle between z axis and line from the tx to rx is lower than ProbeDirectivityCosLimit
// note that this incurs approx. 0.83x  performance pentality, so unless you are sure you need it, it's best to be left disabled
#define RESPECT_DIRCOSLIMIT


// define the initial spread of the simplex for the CUDA-ized simples minimalisation procedures
// used in polyfit_classicMinSearch and polyfit_LockStepMinSearch
/*! \brief Spread for the initial simplex
 *
 * This spreads initial simplex vertices so that the problem space gets explored
 */
#define classicMinSearch_spread 1e-3 


// this will allow nx!=1 when storing scene settings - use with care
#define ALLOW_NX_NONUNITY    

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=
// == system wide re-useable data structures
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=

#define pi 3.14159265358979323846f

struct sCFParam
{
    float x1,y1,z1,x2,y2,z2; // position of source and target point
    float slow1,slow2;       // slowness of wave in medium 1 and 2
    float c0,c1,c2,c3,c4,c5,c6,c7,c8,c9,c10,c11,c12,c13,c14; // parameters of surface for parametric surface    
};


// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=
// == Probe description data
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=


// this is a table that gets alocated in GPU constant memory, meaning fast access to these
__constant__ float constant_ProbeElementLocations[3*max_tx]; 

static unsigned int persistent_ProbeElementLocations_buflength=0; // element locations are still needed to calculate dir-cosine 
static float *persistent_ProbeElementLocations=NULL; // for CPU workspace storage

static unsigned int persistent_ProbeElementCount=0; // number of elements in array.

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=
// == Source data tables and metadata
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=

static float persistent_FMCSamplingRate;
static float persistent_FMCTimeStart;
static float persistent_ProbeDirectivityCosLimit; // parameter for TFM algorithm
static float persistent_ProbeDirectivityCosPower; // parameter for FMC generation

static unsigned int persistent_TxRxList_length=0; // number of A-scans in FMC. Does not have to match number of elements in the array.
static unsigned int persistent_t_idx_max=0; // sample count of FMC data in each line

// area to store raw FMC Data. 
static float *persistent_multidevice_FMCData[max_devicecount]; 
static unsigned int persistent_FMCDataSizeAllocated[max_devicecount]; // indicates how much data has been allocated, 0 if unallocated, check each time to verify that correct amount is allocated
// description of a-scans
static unsigned char *persistent_multidevice_TxRxList[max_devicecount]; 
static unsigned int persistent_TxRxListSizeAllocated[max_devicecount];
// !! use constant memory symbol to access TxRxList
__constant__ unsigned char constant_TxRxList[2*max_tx*max_tx];

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=
// == Output image data and metadata
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=


static float persistent_x0,persistent_y0,persistent_z0;
static float persistent_dx,persistent_dy,persistent_dz;
static unsigned int persistent_nx,persistent_ny,persistent_nz;

static float *persistent_multidevice_TFMData[max_devicecount]; // pointers to buffers for persistent TFM images to stay on GPU
static unsigned int persistent_TFMDataSizeAllocated[max_devicecount];  // indicates how much data has been allocated, 0 if unallocated, check each time to verify that correct amount is allocated

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// Surface data
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=

static float persistent_c0, persistent_c1, persistent_c2, persistent_c3, persistent_c4, persistent_c5, persistent_c6, persistent_c7, persistent_c8, persistent_c9, persistent_c10, persistent_c11, persistent_c12, persistent_c13, persistent_c14;

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// GPU related performance configuration options
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=

static float persistent_performance_lastKernelTime;
static unsigned int persistent_performance_xblocksize=256;
static unsigned int persistent_performance_yblocksize=1; // also governs number of coeff lines loaded by the thread block when pre-caching coefficients

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Coeff algorithm data - consumers
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=

static float *persistent_multidevice_RefractionCoeffData[max_devicecount];
static unsigned int persistent_RefractionCoeffDataSizeAllocated[max_devicecount]; // to check for correct allocation on each GPU 

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Coeff algorithm data - generators
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 

/*! \brief persistent surface type specifier
 *
 * 0 for no refraction
 * 1 for flat z=0
 * 2 for an x-centered cylinder - Tim's case
 * 7 - dual/split polynomial surface, extruded in X direction, and Y is the coordinate for the polynomials. 
 */
static int coeffGeneratorSurfaceId=0;


//static float *persistent_HostCoeffBuffer; // for testing only, to be removed later

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Misc configuration, internal state keeping, and debugging
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 


// MEX call style option
// Object caller places object pointer as first argument, then firstArgument=1;  (points to 2nd argument of the call)
// if the caller is direct, then methodID is the first argument and firstArgument=0; ( points to 1st argument)
static int persistent_firstArgument=0; // used to indicate if the caller is direct or object. 

static int persistIsInited=0;
static int persistent_params_stored=0; // in particular, image size and probe elements have been stored
static int persistent_coeffs_stored=0; // coeff table has been either stored or generated
static int persistent_deviceCount=0;
static int persistent_verbosemode=0;
static int persistent_debugvariable1=0;



// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Local libraries
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 

#include "util\cuda_error_check.cu"
#include "util\v4007_method_freeDeviceMemory.cu"
#include "util\v4007_mexExitFunctionHere.cu"
#include "util\v4007_method_query_CUDA.cu"

#include "util\v4007_method_setDebugVariables.cu"
#include "util\v4007_method_get_performance_counter.cu"

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == TFM Methods
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 

// note that v4007_method_store_params.cu stores image params, probe params, and allocates TFM image buffer. Coeffs are uploaded using method_uploadCoeffs
#include "TFM\v4007_method_store_scene_settings.cu"
#include "TFM\v4007_method_uploadFMC.cu"
#include "TFM\v4007_kernel_RenderTFMImage.cu"
#include "TFM\v4007_method_RenderTFMImage.cu"
#include "TFM\v4007_method_downloadImage.cu"


#include "TFM\v4007_method_setGPUPerformanceOptions.cu"
#include "TFM\v4007_method_uploadCoeffs.cu"


// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Coeff generator Methods
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 

#include "CoeffGenerator\v4007_method_downloadCoeffs.cu"
#include "CoeffGenerator\polyfit_classicMinSearch.cu"
#include "CoeffGenerator\polyfit_LockStepMinSearch.cu"
#include "CoeffGenerator\polyfit_helper.cu"
#include "CoeffGenerator\polyfit17x3.cu"
#include "CoeffGenerator\polyfit17x4.cu"
#include "CoeffGenerator\polyfit17x5.cu"
#include "CoeffGenerator\polyfit17x6.cu"
#include "CoeffGenerator\polyfit17x7.cu"

// include surface functions and CF (CostFunction) Launchers
#include "CoeffGenerator\polyfit_FlatZ.cu"
#include "CoeffGenerator\polyfit_CylnX.cu"
#include "CoeffGenerator\polyfit_SinX.cu"
#include "CoeffGenerator\polyfit_Poly5.cu"
#include "CoeffGenerator\polyfit_DualPolySurface.cu"

// method to evaluate what the points on the surface are
#include "CoeffGenerator\GetSurface.cu"

#include "CoeffGenerator\ind2sub.cu"
#include "CoeffGenerator\v4007_kernel_GenerateTimePoints.cu"
#include "CoeffGenerator\v4007_method_GenerateTimePoints.cu"
#include "CoeffGenerator\v4007_method_GenerateCoeffs.cu"

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == FMC generator
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
#include "FMCSim\v4007_method_ResetFMC.cu"
#include "FMCSim\v4007_method_DownloadFMC.cu"

#include "FMCSim\v4007_kernel_GenerateFMC.cu"
#include "FMCSim\v4007_method_GenerateFMC.cu"

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Basic Licence check
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
#include "cuTFMv4007_LicenceCheck.cu"

       
 
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Some old documentation that i am attached to
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 

// cuTFM: the code kernel that runs on the graphics card
// it is called in "in parallel" for each output pixel.
// note that the output size is assumed to be aligned with the block dimensions!
// this is a strong assumption but i have made it for performance reasons 
//- it is faster to calculate a bit more pixels than for each pixel check 
// if it should be calculated or not. It saves on thread diveregence.
// parameters: 
// x0,y0 - starting coordinates of the result image. 
// dx,dy - spatial steps of the result image (distance between pixels)
// nx,ny - number of pixels in the result image(size of the output array)
//          the pixel coordinates to create image for are calculated 
//          on the fly depending on the built-in thread identifiers 

// NOTE: NAMING CONVENTION
// "*TX" means "current transmiting element of the probe" 
// "*RX" means "current receiving element of the probe"
// "i*" means integer iterator or counter
// "n*" means iteration limit/total count
// "d*" means distance/step size/resoulution
// "*x" and "*y" means on x,y axis respectively

// note that all values are floats(single precision). 
// doubles are supported on newer cards, but they are much slower! 
// single precision should be precise enough for our purpose 
// - 23 bits of precision+sign, 8bits of expotent makes "32-bit single"


// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == mexFunction
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 

// mexFunction: Entry point for matlab. This is what matlab calls.
// for complete and exhaustive documentation see 
// http://www.mathworks.com/support/tech-notes/1600/1605.html#mexFunction

/*! \fn void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
 * \brief Main entry point for calling from Matlab
 * decides which of the included methods to call and passes controll to selected one
 */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
  int methodID;
  int deviceIdx;  
  // attempt to call Matlab function to check if the program is OK to run
       
  // register exit funciton
   mexAtExit(mexExitFunctionHere);
   
  // check if this is first run
   if (persistIsInited==0)
   {
       LicenceCheck();
       mexPrintf("Starting cuTFMv4007 kernel. Jerzy Dziewierz, CUE 2008-2012\n");
       // initialize persistent_multidevice_FMCdata to nulls
       hipGetDeviceCount(&persistent_deviceCount); 
       mexPrintf("%d GPUs detected.\n",persistent_deviceCount); 
       if(persistent_deviceCount == 0){
            mexErrMsgTxt("The current version of this code will fail without any GPUs available.\n");
       }
       // Jurek, I've added the above line for debugging purposes. Feel free to remove.
#ifdef DEBUG_FORCESINGLEGPU       
       //DEBUG CODE: FORCE DeviceCount to 1
       persistent_deviceCount=1;
#endif
       for(deviceIdx=FIRSTDEVICE;deviceIdx<persistent_deviceCount;deviceIdx++)
       {
          persistent_multidevice_FMCData[deviceIdx]=NULL;          
       }       
       persistIsInited=1;
   }
   else
   {
       // nothing special to do if not a first run
   }

// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Process the inputs
// == *=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*=*= 
// == Note that there is a change in the way this mex is called when compared to previous version
// == in this version, in order to support calling the mex as a method of an object, the first argument 
// == is expected to be either object (which is natural for functions called from an object) 
// == OR at least a structure that has neccesary properties that this mex requires.    
// == Some legacy code is left here to allow debugging without object present, but this will be modified to object only later on.
   
   // check if there is more than 0 arguments
   if (nrhs==0)
       mexErrMsgTxt("at least 1 input (methodID) required. Try cuTFMv4007(int32(1)) ");
   
   // check if 1st argument is 1x1 uint32, not complex etc.      
   if( !mxIsNumeric(prhs[0]) || mxIsComplex(prhs[0]) ||
            mxGetN(prhs[0]) * mxGetM(prhs[0])!=1 ) 
                { // the first argument is not the MethodID - check 2nd argument
       
                     if (nrhs==1)
                           mexErrMsgTxt("the first parameter is not methodID and there is no 2nd parameter. At least 1 input (methodID) required. Try cuTFMv4007(int32(1)) ");  
                                        
                    // check if argument 2 (index=1) fits the format of methodID
                        if( mxIsNumeric(prhs[1]) && !mxIsComplex(prhs[1]) &&  mxGetN(prhs[1])==1 && mxGetM(prhs[1])==1 ) 
                        {
                         // seems OK
                          persistent_firstArgument=1;
                        }
                }
   else // meaning the 1st argument DOES match the template of methodID
   {
       persistent_firstArgument=0;
   }   
   // ok, get the MethodID value and act upon it
   methodID=(int)mxGetScalar(prhs[persistent_firstArgument]);
   switch (methodID)
   {
     case 0:  mexPrintf("method 0 called - no output\n"); break;
     case 1:  method_query_CUDA(); break;
     case 17: method_get_performance_counter(nlhs,plhs); break;
     
     // note that the below methods are not compatible with v4006, so they have to have higher MethodID numbers
     // general methods
     
     case 19:  method_store_scene_settings(nrhs,prhs); break;     
     case 20:  method_uploadFMC(nrhs,prhs); break;
     case 21:  method_RenderTFMImage(); break;
     case 22:  method_downloadImage(nlhs,plhs); break; 
     case 23:  method_setDebugVariables(nlhs,plhs,nrhs,prhs); break; 
     case 24:  method_setGPUPerformanceOptions(nrhs,prhs); break; 
     
     // coeff generator
     case 25:  method_uploadCoeffs(nrhs,prhs); break;
     case 26:  method_GenerateCoeffs(nlhs,plhs,nrhs,prhs); break;
     case 27:  method_downloadCoeffs(nlhs,plhs); break;
     
     // FMC generator
     case 28: method_ResetFMC(nlhs,plhs,nrhs,prhs); break;
     case 29: method_DownloadFMC(nlhs,plhs,nrhs,prhs); break;
     case 30: method_GenerateFMC(nlhs,plhs,nrhs,prhs); break;
     
     // evaluate surface
     case 31: method_GetSurface(nlhs,plhs,nrhs,prhs); break;
     
     
     // case28: method_doAbsLog(nlhs,plhs);
               
     default: mexPrintf("method %d not implemented or invalid\n",methodID); break;
     
   }
}
 